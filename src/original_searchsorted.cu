
#include <hip/hip_runtime.h>
#include <iostream>
#include <climits>
#include <assert.h>

__device__  __host__
int midpoint(int a, int b)
{
    return a + (b-a)/2;
}

__device__ __host__
int eval(int A[], int i, int val, int imin, int imax)
{

    int low = (A[i] <= val);
    int high = (A[i+1] > val);

    if (low && high) {
        return 0;
    } else if (low) {
        return -1;
    } else {
        return 1;
    }
}

__device__ __host__
int binary_search(int A[], int val, int imin, int imax)
{
    while (imax >= imin) {
        int imid = midpoint(imin, imax);
        int e = eval(A, imid, val, imin, imax);
        if(e == 0) {
            return imid;
        } else if (e < 0) {
            imin = imid;
        } else {
            imax = imid;
        }
    }

    return -1;
}


__device__ __host__
int linear_search(int A[], int val, int imin, int imax)
{
    int res = -1;
    for(int i=imin; i<(imax-1); i++) {
        if (A[i+1] > val) {
            res = i;
            break;
        }
    }

    return res;
}

template<int version>
__global__
void search(int * source, int * result, int Nin, int Nout)
{
    extern __shared__ int buff[];
    int tid = threadIdx.x + blockIdx.x*blockDim.x;

    int val = INT_MAX;
    if (tid < Nin) val = source[threadIdx.x];
    buff[threadIdx.x] = val;
    __syncthreads();

    int res;
    switch(version) {

        case 0:
        res = binary_search(buff, threadIdx.x, 0, blockDim.x);
        break;

        case 1:
        res = linear_search(buff, threadIdx.x, 0, blockDim.x);
        break;
    }

    if (tid < Nout) result[tid] = res;
}

int main(void)
{
    const int inputLength = 128000;
    const int isize = inputLength * sizeof(int);
    const int outputLength = 256;
    const int osize = outputLength * sizeof(int);

    int * hostInput = new int[inputLength];
    int * hostOutput = new int[outputLength];
    int * deviceInput;
    int * deviceOutput;

    for(int i=0; i<inputLength; i++) {
        hostInput[i] = -200 + 5*i;
    }

    hipMalloc((void**)&deviceInput, isize);
    hipMalloc((void**)&deviceOutput, osize);

    hipMemcpy(deviceInput, hostInput, isize, hipMemcpyHostToDevice);

    dim3 DimBlock(256, 1, 1);
    dim3 DimGrid(1, 1, 1);
    DimGrid.x = (outputLength / DimBlock.x) +
                ((outputLength % DimBlock.x > 0) ? 1 : 0);
    size_t shmsz = DimBlock.x * sizeof(int);

    for(int i=0; i<5; i++) {
        search<1><<<DimGrid, DimBlock, shmsz>>>(deviceInput, deviceOutput,
                inputLength, outputLength);
    }

    for(int i=0; i<5; i++) {
        search<0><<<DimGrid, DimBlock, shmsz>>>(deviceInput, deviceOutput,
                inputLength, outputLength);
    }

    hipMemcpy(hostOutput, deviceOutput, osize, hipMemcpyDeviceToHost);

    for(int i=0; i<outputLength; i++) {
        int idx = hostOutput[i];
        int tidx = i % DimBlock.x;
        assert( (hostInput[idx] <= tidx) && (tidx < hostInput[idx+1]) );
    }
    hipDeviceReset();

    return 0;
}
