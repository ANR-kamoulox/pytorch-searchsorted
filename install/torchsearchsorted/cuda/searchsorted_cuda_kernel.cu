#include "hip/hip_runtime.h"
#include <iostream>
#include <climits>
#include <assert.h>
#include <hiprand.h>
#include <stdio.h>
#include <math.h>
#include <float.h>




#include "searchsorted_cuda_kernel.h"

#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

template <typename scalar_t>
__device__
int eval(scalar_t val, scalar_t *a, int row, int col, int ncol)
{
  /* Evaluates whether a[row,col] < val <= a[row, col+1]*/

    if (col == ncol-1){
      // we are on the right border. This is the answer.
      return 0;}

    // a[row,col] <= val ?
    int is_lower = (a[row*ncol + col] < val);

    // a[row,col+1] > val ?
    int is_next_higher = (a[row*ncol + col + 1] >= val);

    if (is_lower && is_next_higher) {
      // we found the answer
        return 0;
    } else if (is_lower) {
      // answer is on the right side
        return 1;
    } else {
      // answer is on the left side
        return -1;
    }
}

template <typename scalar_t>
__device__
int binary_search(scalar_t *a, int row, scalar_t val, int ncol)
{
  /* Look for the value `val` within row `row` of matrix `a`, which
  has `ncol` columns.

  the `a` matrix is assumed sorted in increasing order, row-wise

  Returns -1 if `val` is smaller than the smallest value found within that
  row of `a`. Otherwise, return the column index `res` such that:
  a[row, col] < val <= a[row, col+1]. in case `val` is larger than the
  largest element of that row of `a`, simply return `ncol`-1. */

  //start with left at 0 and right at ncol
  int right = ncol;
  int left = 0;

  while (right >= left) {
      // take the midpoint of current left and right cursors
      int mid = left + (right-left)/2;

      // check the relative position of val: is this midpoint smaller or larger
      // than val ?
      int rel_pos = eval<scalar_t>(val, a, row, mid, ncol);

      // we found the point
      if(rel_pos == 0) {
          return mid;
      } else if (rel_pos > 0) {
        // the answer is on the right side
          left = mid;
      } else {
        // the answer is on the left side
        if (!mid)
        {
          //if we're already on the first element, we didn't find
          return -1;}
        else
        {right = mid;}
      }
  }
  return -1;
}

template <typename scalar_t>
__global__
void searchsorted_kernel(
  scalar_t *res,
  scalar_t *a,
  scalar_t *v,
  int nrow_res, int nrow_a, int nrow_v, int ncol_a, int ncol_v)
{
    // get current row and column
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    // check whether we are outside the bounds of what needs be computed.
    if ((row >= nrow_res) || (col >= ncol_v)) {
      return;}

    // get the value to look for
    int row_in_v = (nrow_v==1) ? 0: row;
    int row_in_a = (nrow_a==1) ? 0: row;
    int idx_in_v = row_in_v*ncol_v+col;
    int idx_in_res = row*ncol_v+col;

    // apply binary search
    res[idx_in_res] = binary_search(a, row_in_a, v[idx_in_v], ncol_a)+1;
}


void searchsorted_cuda(
  at::Tensor a,
  at::Tensor v,
  at::Tensor res){

      // Get the dimensions
      auto nrow_a = a.size(/*dim=*/0);
      auto nrow_v = v.size(/*dim=*/0);
      auto ncol_a = a.size(/*dim=*/1);
      auto ncol_v = v.size(/*dim=*/1);

      auto nrow_res = std::max(nrow_a, nrow_v);

      // prepare the kernel configuration
      dim3 threads(ncol_v, nrow_res);
      dim3 blocks(1, 1);
      if (nrow_res*ncol_v > 1024){
         threads.x = std::min(1024, int(ncol_v));
         threads.y = floor(1024/threads.x);
         blocks.x = ceil(double(ncol_v)/double(threads.x));
         blocks.y = ceil(double(nrow_res)/double(threads.y));
      }

      AT_DISPATCH_ALL_TYPES(res.type(), "searchsorted cuda", ([&] {
        searchsorted_kernel<scalar_t><<<blocks, threads>>>(
          res.data<scalar_t>(),
          a.data<scalar_t>(),
          v.data<scalar_t>(),
          nrow_res, nrow_a, nrow_v, ncol_a, ncol_v);
      }));

  }
